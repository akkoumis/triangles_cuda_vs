#include <hipsparse.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>

#define CLEANUP(s)                                   \
do {                                                 \
    printf ("%s\n", s);                              \
    if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);\
    if (cooColIndexHostPtr) free(cooColIndexHostPtr);\
    if (cooValHostPtr)      free(cooValHostPtr);     \
if (cooRowIndex)        hipFree(cooRowIndex);   \
            	printf("TEST\n");	\
if (cooColIndex)        hipFree(cooColIndex);   \
if (cooVal)             hipFree(cooVal);        \
    if (csrRowPtr)          hipFree(csrRowPtr);     \
	if (descr)              hipsparseDestroyMatDescr(descr);\
    if (handle)             hipsparseDestroy(handle); \
	hipDeviceReset();          \
    fflush (stdout);                                 \
} while (0)


int main()
{
	hipError_t cudaStat1, cudaStat2, cudaStat3;
	hipsparseStatus_t status;
	hipsparseHandle_t handle = 0;
	hipsparseMatDescr_t descr = 0, descrA2 = 0;

	//int NNZ = 6629222;	// auto
	//int sizeOfMatrix = 448695;
	int NNZ = 16313034; // britain
	int sizeOfMatrix = 7733822;
	//int NNZ = 25165738;	// delaunay
	//int sizeOfMatrix = 4194304;

	int *cooRowIndexHostPtr = 0;
	int *cooColIndexHostPtr = 0;
	float *cooValHostPtr = 0;
	int *cooRowIndex = 0;
	int *cooColIndex = 0;
	float *cooVal = 0;
	int *csrRowPtr = 0;
	int *csrRowPtrA2 = 0;
	int *csrColIndA2 = 0;
	float *csrValA2 = 0;

	cooRowIndexHostPtr = (int*)malloc(NNZ * sizeof(cooRowIndexHostPtr[0]));
	cooColIndexHostPtr = (int*)malloc(NNZ * sizeof(cooColIndexHostPtr[0]));
	cooValHostPtr = (float*)malloc(NNZ * sizeof(cooValHostPtr[0]));
	if ((!cooRowIndexHostPtr) || (!cooColIndexHostPtr) || (!cooValHostPtr)) {
		CLEANUP("Host malloc failed (matrix)");
		return 1;
	}

	FILE* fp;
	int buff_int = 0;

	printf("Scanning file...\n");
	//fp = fopen("auto_A.txt", "r");	//auto
	fp = fopen("britain_A.txt", "r");	//britain
	//fp = fopen("delaunay_A.txt", "r");	//delaunay

	for (int i = 0; i < NNZ; i++)
	{
		fscanf(fp, "%d", &cooRowIndexHostPtr[i]);
		fscanf(fp, "%d", &cooColIndexHostPtr[i]);
		fscanf(fp, "%f", &cooValHostPtr[i]);
	}
	fclose(fp);

	printf("Scan completed!\n");
	printf("%d\t%d\t%.2f\n", cooRowIndexHostPtr[NNZ - 1], cooColIndexHostPtr[NNZ - 1], cooValHostPtr[NNZ - 1]);

	cudaStat1 = hipMalloc((void**)& cooRowIndex, NNZ * sizeof(cooRowIndex[0]));
	cudaStat2 = hipMalloc((void**)& cooColIndex, NNZ * sizeof(cooColIndex[0]));
	cudaStat3 = hipMalloc((void**)& cooVal, NNZ * sizeof(cooVal[0]));
	if ((cudaStat1 != hipSuccess) ||
		(cudaStat2 != hipSuccess) ||
		(cudaStat3 != hipSuccess) ) {
		CLEANUP("Device malloc failed");
		return 1;
	}
	cudaStat1 = hipMemcpy(cooRowIndex, cooRowIndexHostPtr,
		(size_t)(NNZ * sizeof(cooRowIndex[0])),
		hipMemcpyHostToDevice);
	cudaStat2 = hipMemcpy(cooColIndex, cooColIndexHostPtr,
		(size_t)(NNZ * sizeof(cooColIndex[0])),
		hipMemcpyHostToDevice);
	cudaStat3 = hipMemcpy(cooVal, cooValHostPtr,
		(size_t)(NNZ * sizeof(cooVal[0])),
		hipMemcpyHostToDevice);
	if ((cudaStat1 != hipSuccess) ||
		(cudaStat2 != hipSuccess) ||
		(cudaStat3 != hipSuccess) ) {
		CLEANUP("Memcpy from Host to Device failed");
		return 1;
	}

	/* initialize cusparse library */ 
	status = hipsparseCreate(&handle);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		CLEANUP("CUSPARSE Library initialization failed");
		return 1;
	}

	/* create and setup matrix descriptor */
	status = hipsparseCreateMatDescr(&descr);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		CLEANUP("Matrix descriptor initialization failed");
		return 1;
	}
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ONE);

	cudaStat1 = hipMalloc((void**)& csrRowPtr, ((size_t)sizeOfMatrix + 1) * sizeof(csrRowPtr[0]));
	if (cudaStat1 != hipSuccess) {
		CLEANUP("Device malloc failed (csrRowPtr)");
		return 1;
	}
	status = hipsparseXcoo2csr(handle, cooRowIndex, NNZ, sizeOfMatrix,
		csrRowPtr, HIPSPARSE_INDEX_BASE_ONE);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		CLEANUP("Conversion from COO to CSR format failed");
		return 1;
	}
	printf("A converted to CSR.\n");


	/*int devId;
	hipDeviceProp_t prop;
	hipError_t cudaStat;
	cudaStat = hipGetDevice(&devId);
	if (hipSuccess != cudaStat) {
		CLEANUP("hipGetDevice failed");
		printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
		return 1;
	}
	cudaStat = hipGetDeviceProperties(&prop, devId);
	if (hipSuccess != cudaStat) {
		CLEANUP("hipGetDeviceProperties failed");
		printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
		return 1;
	}*/

	
	// assume matrices A, B and D are ready.
	int baseC, nnzC;
	csrgemm2Info_t info = NULL;
	size_t bufferSize;
	void* buffer = NULL;
	// nnzTotalDevHostPtr points to host memory
	int* nnzTotalDevHostPtr = &nnzC;
	float alpha = 1.0;
	float beta = 0.0;
	hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
	printf("TEST\n");
	// step 1: create an opaque structure
	hipsparseCreateCsrgemm2Info(&info);
	printf("TEST1\n");

	// step 2: allocate buffer for csrgemm2Nnz and csrgemm2
	hipsparseScsrgemm2_bufferSizeExt(handle, sizeOfMatrix, sizeOfMatrix, sizeOfMatrix, &alpha,
		descr, NNZ, csrRowPtr, cooColIndex,
		descr, NNZ, csrRowPtr, cooColIndex,
		&beta,
		descr, NNZ, csrRowPtr, cooColIndex,
		info,
		&bufferSize);
	hipMalloc(&buffer, bufferSize);
	printf("TEST2\n");

	// step 3: compute csrRowPtrC
	hipMalloc((void**)& csrRowPtrA2, sizeof(int)* ((size_t)sizeOfMatrix + 1));
	printf("TEST3\n");
	/*hipsparseXcsrgemm2Nnz(handle, sizeOfMatrix, sizeOfMatrix, sizeOfMatrix,
		descr, NNZ, csrRowPtr, cooColIndex,
		descr, NNZ, csrRowPtr, cooColIndex,
		descr, NNZ, csrRowPtr, cooColIndex,
		descr, csrRowPtrA2, nnzTotalDevHostPtr,
		info, buffer);
	*/


	if (NULL != nnzTotalDevHostPtr) {
		nnzC = *nnzTotalDevHostPtr;
	}
	else {
		hipMemcpy(&nnzC, csrRowPtrA2 + sizeOfMatrix, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&baseC, csrRowPtrA2, sizeof(int), hipMemcpyDeviceToHost);
		nnzC -= baseC;
	}

	nnzC = 27968324;

	// step 4: finish sparsity pattern and value of C
	hipMalloc((void**)& csrColIndA2, sizeof(int)* nnzC);
	hipMalloc((void**)& csrValA2, sizeof(double)* nnzC);
	// Remark: set csrValC to null if only sparsity pattern is required.
	printf("TEST4\n");
	hipsparseScsrgemm2(handle, sizeOfMatrix, sizeOfMatrix, sizeOfMatrix, &alpha,
		descr, NNZ, cooVal, csrRowPtr, cooColIndex,
		descr, NNZ, cooVal, csrRowPtr, cooColIndex,
		&beta,
		descr, NNZ, cooVal, csrRowPtr, cooColIndex,
		descr, csrValA2, csrRowPtrA2, csrColIndA2,
		info, buffer);

	// step 5: destroy the opaque structure
	hipsparseDestroyCsrgemm2Info(info);





	/* destroy matrix descriptor */
	status = hipsparseDestroyMatDescr(descr);
	descr = 0;
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		CLEANUP("Matrix descriptor destruction failed");
		return 1;
	}

	/* destroy handle */
	status = hipsparseDestroy(handle);
	handle = 0;
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		CLEANUP("CUSPARSE Library release of resources failed");
		return 1;
	}

	//CLEANUP("The end.\n");

	if (cooRowIndexHostPtr) free(cooRowIndexHostPtr); 
	if (cooColIndexHostPtr) free(cooColIndexHostPtr); 
	if (cooValHostPtr)      free(cooValHostPtr);     
	if (cooRowIndex)        hipFree(cooRowIndex);   
	printf("TEST\n");
	if (cooColIndex)        hipFree(cooColIndex);   
	if (cooVal)             hipFree(cooVal);        
	if (csrRowPtr)          hipFree(csrRowPtr);     
	if (descr)              hipsparseDestroyMatDescr(descr); 
	if (handle)             hipsparseDestroy(handle); 
	hipDeviceReset();

}
