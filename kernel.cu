#include <hipsparse.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>

#define CLEANUP(s)                                   \
do {                                                 \
    printf ("%s\n", s);                              \
    if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);\
    if (cooColIndexHostPtr) free(cooColIndexHostPtr);\
    if (cooValHostPtr)      free(cooValHostPtr);     \
if (cooRowIndex)        hipFree(cooRowIndex);   \
            	printf("TEST\n");	\
if (cooColIndex)        hipFree(cooColIndex);   \
if (cooVal)             hipFree(cooVal);        \
    if (csrRowPtr)          hipFree(csrRowPtr);     \
	if (descr)              hipsparseDestroyMatDescr(descr);\
    if (handle)             hipsparseDestroy(handle); \
	hipDeviceReset();          \
    fflush (stdout);                                 \
} while (0)


int main()
{
	hipError_t cudaStat1, cudaStat2, cudaStat3;
	hipsparseStatus_t status;
	hipsparseHandle_t handle = 0;
	hipsparseMatDescr_t descr = 0, descrA2 = 0;

	//int NNZ = 6629222;	// auto
	//int sizeOfMatrix = 448695;
	int NNZ = 16313034; // britain
	int sizeOfMatrix = 7733822;
	//int NNZ = 25165738;	// delaunay
	//int sizeOfMatrix = 4194304;

	int *cooRowIndexHostPtr = 0;
	int *cooColIndexHostPtr = 0;
	float *cooValHostPtr = 0;
	int *cooRowIndex = 0;
	int *cooColIndex = 0;
	float *cooVal = 0;
	int *csrRowPtr = 0;
	int *csrRowPtrA2 = 0;
	int *csrColIndA2 = 0;
	float *csrValA2 = 0;

	cooRowIndexHostPtr = (int*)malloc(NNZ * sizeof(cooRowIndexHostPtr[0]));
	cooColIndexHostPtr = (int*)malloc(NNZ * sizeof(cooColIndexHostPtr[0]));
	cooValHostPtr = (float*)malloc(NNZ * sizeof(cooValHostPtr[0]));
	if ((!cooRowIndexHostPtr) || (!cooColIndexHostPtr) || (!cooValHostPtr)) {
		CLEANUP("Host malloc failed (matrix)");
		return 1;
	}

	FILE* fp;
	int buff_int = 0;

	printf("Scanning file...\n");
	//fp = fopen("auto_A.txt", "r");	//auto
	fp = fopen("britain_A.txt", "r");	//britain
	//fp = fopen("delaunay_A.txt", "r");	//delaunay

	for (int i = 0; i < NNZ; i++)
	{
		fscanf(fp, "%d", &cooRowIndexHostPtr[i]);
		fscanf(fp, "%d", &cooColIndexHostPtr[i]);
		fscanf(fp, "%f", &cooValHostPtr[i]);
	}
	fclose(fp);

	printf("Scan completed!\n");
	printf("%d\t%d\t%.2f\n", cooRowIndexHostPtr[NNZ - 1], cooColIndexHostPtr[NNZ - 1], cooValHostPtr[NNZ - 1]);

	cudaStat1 = hipMalloc((void**)& cooRowIndex, NNZ * sizeof(cooRowIndex[0]));
	cudaStat2 = hipMalloc((void**)& cooColIndex, NNZ * sizeof(cooColIndex[0]));
	cudaStat3 = hipMalloc((void**)& cooVal, NNZ * sizeof(cooVal[0]));
	if ((cudaStat1 != hipSuccess) ||
		(cudaStat2 != hipSuccess) ||
		(cudaStat3 != hipSuccess) ) {
		CLEANUP("Device malloc failed");
		return 1;
	}
	cudaStat1 = hipMemcpy(cooRowIndex, cooRowIndexHostPtr,
		(size_t)(NNZ * sizeof(cooRowIndex[0])),
		hipMemcpyHostToDevice);
	cudaStat2 = hipMemcpy(cooColIndex, cooColIndexHostPtr,
		(size_t)(NNZ * sizeof(cooColIndex[0])),
		hipMemcpyHostToDevice);
	cudaStat3 = hipMemcpy(cooVal, cooValHostPtr,
		(size_t)(NNZ * sizeof(cooVal[0])),
		hipMemcpyHostToDevice);
	if ((cudaStat1 != hipSuccess) ||
		(cudaStat2 != hipSuccess) ||
		(cudaStat3 != hipSuccess) ) {
		CLEANUP("Memcpy from Host to Device failed");
		return 1;
	}

	/* initialize cusparse library */ 
	status = hipsparseCreate(&handle);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		CLEANUP("CUSPARSE Library initialization failed");
		return 1;
	}

	/* create and setup matrix descriptor */
	status = hipsparseCreateMatDescr(&descr);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		CLEANUP("Matrix descriptor initialization failed");
		return 1;
	}
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ONE);

	cudaStat1 = hipMalloc((void**)& csrRowPtr, ((size_t)sizeOfMatrix + 1) * sizeof(csrRowPtr[0]));
	if (cudaStat1 != hipSuccess) {
		CLEANUP("Device malloc failed (csrRowPtr)");
		return 1;
	}
	status = hipsparseXcoo2csr(handle, cooRowIndex, NNZ, sizeOfMatrix,
		csrRowPtr, HIPSPARSE_INDEX_BASE_ONE);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		CLEANUP("Conversion from COO to CSR format failed");
		return 1;
	}
	printf("A converted to CSR.\n");


	/*int devId;
	hipDeviceProp_t prop;
	hipError_t cudaStat;
	cudaStat = hipGetDevice(&devId);
	if (hipSuccess != cudaStat) {
		CLEANUP("hipGetDevice failed");
		printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
		return 1;
	}
	cudaStat = hipGetDeviceProperties(&prop, devId);
	if (hipSuccess != cudaStat) {
		CLEANUP("hipGetDeviceProperties failed");
		printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
		return 1;
	}*/

	//int *cooColIndTemp = cooC
	int baseC, nnzC;
	// nnzTotalDevHostPtr points to host memory
	int* nnzTotalDevHostPtr = &nnzC;
	hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
	hipMalloc((void**)& csrRowPtrA2, sizeof(int)* ((size_t)sizeOfMatrix + 1));
	status = hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		sizeOfMatrix, sizeOfMatrix, sizeOfMatrix,
		descr, NNZ, csrRowPtr, cooColIndex,
		descr, NNZ, csrRowPtr, cooColIndex,
		descr, csrRowPtrA2, nnzTotalDevHostPtr);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("Error finding nnz = %d\n",status);
	}
	printf("\nTEST\n");
	if (NULL != nnzTotalDevHostPtr) {
		nnzC = *nnzTotalDevHostPtr;
		printf("NNZ of A2 = %d\n", nnzC);
	}
	else {
		hipMemcpy(&nnzC, csrRowPtrA2 + sizeOfMatrix, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&baseC, csrRowPtrA2, sizeof(int), hipMemcpyDeviceToHost);
		nnzC -= baseC;
	}
	hipMalloc((void**)& csrColIndA2, sizeof(int)* nnzC);
	hipMalloc((void**)& csrValA2, sizeof(float)* nnzC);
	hipsparseScsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		sizeOfMatrix, sizeOfMatrix, sizeOfMatrix,
		descr, NNZ,
		cooVal, csrRowPtr, cooColIndex,
		descr, NNZ,
		cooVal, csrRowPtr, cooColIndex,
		descr,
		csrValA2, csrRowPtrA2, csrColIndA2);



	/* destroy matrix descriptor */
	status = hipsparseDestroyMatDescr(descr);
	descr = 0;
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		CLEANUP("Matrix descriptor destruction failed");
		return 1;
	}

	/* destroy handle */
	status = hipsparseDestroy(handle);
	handle = 0;
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		CLEANUP("CUSPARSE Library release of resources failed");
		return 1;
	}

	//CLEANUP("The end.\n");

	if (cooRowIndexHostPtr) free(cooRowIndexHostPtr); 
	if (cooColIndexHostPtr) free(cooColIndexHostPtr); 
	if (cooValHostPtr)      free(cooValHostPtr);     
	if (cooRowIndex)        hipFree(cooRowIndex);   
	printf("TEST\n");
	if (cooColIndex)        hipFree(cooColIndex);   
	if (cooVal)             hipFree(cooVal);        
	if (csrRowPtr)          hipFree(csrRowPtr);     
	if (descr)              hipsparseDestroyMatDescr(descr); 
	if (handle)             hipsparseDestroy(handle); 
	hipDeviceReset();

}
